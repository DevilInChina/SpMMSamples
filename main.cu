
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

void swap(int *a, int *b) {
    if (*a != *b)
        *a ^= *b ^= *a ^= *b;
}

int cmp(const void *a, const void *b) {
    return *((int *) a) - *((int *) b);
}

#define NNZ 50
#define WARMUP_TIMES 5
#define VALUE_TYPE double
#define BENCH_TIMES 10

void GenerateCsr(int **RowPtr, int **ColIdx, int m) {
    srand(m);
    *RowPtr = (int *) malloc(sizeof(int) * (m + 1));
    *ColIdx = (int *) malloc(sizeof(int) * (m * NNZ));

    int *randCol = (int *) malloc(sizeof(int) * m * 2);
    for (int i = 0; i < m; ++i) {
        randCol[i] = i;
    }

    for (int i = 0; i < m; ++i) {
        swap(randCol + i, randCol + rand() % m);
    }
    memcpy(randCol + m, randCol, sizeof(int) * m);
    (*RowPtr)[0] = 0;
    for (int i = 1; i <= m; ++i) {
        int nnz = rand() % NNZ + 1;
        if (nnz > m)nnz = m;
        (*RowPtr)[i] = (*RowPtr)[i - 1] + nnz;
        int buff = rand() % m;
        memcpy(*ColIdx + (*RowPtr)[i - 1], randCol + buff, nnz * sizeof(int));
        qsort(*ColIdx + (*RowPtr)[i - 1], nnz, sizeof(int), cmp);
    }

    *ColIdx = (int *) realloc(*ColIdx, sizeof(int) * (*RowPtr)[m]);
    free(randCol);
}

void GeMM(int m, int width,
          VALUE_TYPE *MatrixVal, VALUE_TYPE *denseRightMatrix,
          VALUE_TYPE *Res, double *time_val) {

    struct timeval t1, t2;
    *time_val = 0;
    for (int _ = 0; _ < BENCH_TIMES; ++_) {
        memset(Res, 0, sizeof(VALUE_TYPE) * width * m);
        gettimeofday(&t1, NULL);
#pragma omp parallel for
        for (int i = 0; i < m; ++i) {
            for (int j = 0; j < m; ++j) {
                for (int k = 0; k < width; ++k) {
                    Res[i * width + k] += MatrixVal[i * m + j] * denseRightMatrix[j * width + k];
                }
            }
        }
        gettimeofday(&t2, NULL);
        *time_val += ((t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0);
    }
    *time_val /= BENCH_TIMES;
}

void csrSpMM(int m, int *RowPtr, int *ColIdx, VALUE_TYPE *CsrVal,
             int width, VALUE_TYPE *denseRightMatrix, VALUE_TYPE *Res, double *time_val) {

    struct timeval t1, t2;
    *time_val = 0;
    for (int _ = 0; _ < BENCH_TIMES; ++_) {
        memset(Res, 0, sizeof(VALUE_TYPE) * width * m);
        gettimeofday(&t1, NULL);
#pragma omp parallel for
        for (int i = 0; i < m; ++i) {
            for (int j = RowPtr[i]; j < RowPtr[i + 1]; ++j) {
                for (int k = 0; k < width; ++k) {
                    Res[i * width + k] += CsrVal[j] * denseRightMatrix[ColIdx[j] * width + k];
                }
            }
        }
        gettimeofday(&t2, NULL);
        *time_val += ((t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0);
    }
    *time_val /= BENCH_TIMES;
}

void compareUndPrint(const char *name, const double *C_Golden, const double *C_ref, int m, int n) {

    int count1 = 0;
    for (int i = 0; i < m * n; i++)
        if (C_Golden[i] != C_ref[i]) {
            //    printf("%d %d %f %f\n",i/n,i%n,C[i],C_golden[i]);
            count1++;
        }
    if (count1 == 0)
        printf("(%s)(row-col, A and B are in row-major) PASS!\n\n", name);
    else
        printf("(%s)(row-col, A and B are in row-major) NOT PASS!\n\n", name);
}

__global__ void SpMMKernel(int m, int *RowPtr, int *ColIdx, VALUE_TYPE *CsrVal,
                           int width, VALUE_TYPE *denseRightMatrix, VALUE_TYPE *Res) {
// Each thread computes one element of C
// by accumulating results into Cvalue
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    //int col = blockIdx.y * blockDim.y + threadIdx.y;
    for(int k = 0 ; k < width ; ++k) {
        Res[row * width + k] = 0;
    }
    for (int j = RowPtr[row]; j < RowPtr[row + 1]; ++j) {
        double val = 0;
        for(int k = 0 ; k < width ; ++k) {
            Res[row * width + k] += CsrVal[j] * denseRightMatrix[ColIdx[j] * width + k];
        }
    }
}


void spMM_cuda_yours(int m, int *RowPtr, int *ColIdx, VALUE_TYPE *CsrVal,
                     int width, VALUE_TYPE *denseRightMatrix, VALUE_TYPE *Res, double *time_value) {

    int *d_RowPtr, *d_ColIdx;

    size_t size = (m + 1) * sizeof(int);
    hipMalloc(&d_RowPtr, size);
    hipMemcpy(d_RowPtr, RowPtr, size,
               hipMemcpyHostToDevice);

    size = RowPtr[m] * sizeof(int);
    hipMalloc(&d_ColIdx, size);
    hipMemcpy(d_ColIdx, ColIdx, size,
               hipMemcpyHostToDevice);
// Allocate C in device memory

    double *d_CsrVal, *d_denseRightMatrix, *d_Res;
    size = RowPtr[m] * sizeof(double);
    hipMalloc(&d_CsrVal, size);
    hipMemcpy(d_CsrVal, CsrVal, size,
               hipMemcpyHostToDevice);

    size = sizeof(double) * m * width;

    hipMalloc(&d_denseRightMatrix, size);
    hipMemcpy(d_denseRightMatrix, denseRightMatrix, size,
               hipMemcpyHostToDevice);

    hipMalloc(&d_Res, size);
    dim3 dimBlock(1, 1);
    dim3 dimGrid(m, 1);

    for (int i = 0; i < WARMUP_TIMES; ++i) {

        ///// edit your warmup code here

        SpMMKernel<<<dimGrid,dimBlock>>>(m,d_RowPtr,d_ColIdx,d_CsrVal,
                                         width,d_denseRightMatrix,d_Res);
        ////
    }
    timeval t1, t2;
    hipDeviceSynchronize();
    *time_value = 0;
    for (int i = 0; i < BENCH_TIMES; ++i) {
        // cublasSgemm('N', 'N', m, n, k, 1.0f, d_A, m, d_B, k, 0, d_C, m);

        gettimeofday(&t1, nullptr);
        ///// edit your code here

        SpMMKernel<<<dimGrid,dimBlock>>>(m,d_RowPtr,d_ColIdx,d_CsrVal,
                                         width,d_denseRightMatrix,d_Res);


        ////

        hipDeviceSynchronize();
        gettimeofday(&t2, nullptr);
        *time_value += (t2.tv_sec - t1.tv_sec) + (t2.tv_usec - t1.tv_usec) / 1000000.0;
    }


    *time_value /= BENCH_TIMES;
    hipMemcpy(Res, d_Res, size,
               hipMemcpyDeviceToHost);

    hipFree(d_ColIdx);
    hipFree(d_Res);
    hipFree(d_CsrVal);
    hipFree(d_RowPtr);
    hipFree(d_denseRightMatrix);

}

int main(int argc, char **argv) {
    if (argc != 3) {
        printf("First parameter is height and width of left matrix.\n"
               "Second parameter is width of right matrix(8,16,32 is recommended).");
        exit(0);
    }
    int m = atoi(argv[1]);
    int width = atoi(argv[2]);
    int *RowPtr, *ColIdx;
    GenerateCsr(&RowPtr, &ColIdx, m);
    VALUE_TYPE *CsrVal = (VALUE_TYPE *) malloc(sizeof(VALUE_TYPE) * (RowPtr[m]));
    for (int i = 0; i < RowPtr[m]; ++i) {
        CsrVal[i] = (rand() % 8 + 1) / 8.0;
    }

    VALUE_TYPE *DenseMatrixVal = (VALUE_TYPE *) malloc(sizeof(VALUE_TYPE) * m * m);
    memset(DenseMatrixVal, 0, sizeof(VALUE_TYPE) * m * m);

    for (int i = 0; i < m; ++i) {
        for (int j = RowPtr[i]; j < RowPtr[i + 1]; ++j) {
            DenseMatrixVal[i * m + ColIdx[j]] = CsrVal[j];
        }
    }

    VALUE_TYPE *RightThinMatrix = (VALUE_TYPE *) malloc(sizeof(VALUE_TYPE) * width * m);
    srand(width);
    for (int i = 0; i < width * m; ++i) {
        RightThinMatrix[i] = rand() % 32 * 0.125;
    }
    VALUE_TYPE *Res_Golden = (VALUE_TYPE *) malloc(sizeof(VALUE_TYPE) * width * m);
    VALUE_TYPE *Res = (VALUE_TYPE *) malloc(sizeof(VALUE_TYPE) * width * m);
    double time_value;

    printf("Matrix A is %i x %i, matrix B is %i x %i\n", m, m, m, width);
    printf("Matrix A has a sparsity of %.3f%%\n", RowPtr[m] * 100.0 / m / m);

    GeMM(m, width, DenseMatrixVal, RightThinMatrix, Res_Golden, &time_value);
    const char *Name = "GeMM";
    printf("\n(%s)(row-col, A and B are in row-major)) used %4.5f ms\n",
           Name, time_value);


    csrSpMM(m, RowPtr, ColIdx, CsrVal, width, RightThinMatrix, Res, &time_value);
    Name = "csrSpMM";
    printf("\n(%s)(row-col, A and B are in row-major)) used %4.5f ms\n",
           Name, time_value);
    compareUndPrint(Name, Res, Res_Golden, m, width);


    spMM_cuda_yours(m, RowPtr, ColIdx, CsrVal, width, RightThinMatrix, Res, &time_value);
    Name = "cudaSpMM";
    printf("\n(%s)(row-col, A and B are in row-major)) used %4.5f ms\n",
           Name, time_value);
    compareUndPrint(Name, Res, Res_Golden, m, width);

    return 0;
}
